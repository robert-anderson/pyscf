/*
  simple wrapper to utility cuda routines  
 */

#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


extern "C" int CountDevices()
{
  int num_gpus = -1;
  checkCudaErrors(hipGetDeviceCount(&num_gpus));
  return num_gpus;
}

extern "C" void SetDevice(int gpu_id)
{
  checkCudaErrors(hipSetDevice(gpu_id));
}

extern "C" int GetDevice()
{
  int gpu_id = -1;
  checkCudaErrors(hipGetDevice(&gpu_id));
  return gpu_id;
}
